#include "hip/hip_runtime.h"
#include<mpi.h>
#include<iostream>
#include<time.h>
#include<Windows.h>
#define N 5
using namespace std;

int graf[N][N] =
{
	0,3,9,8,3,
	5,0,1,4,2,
	6,6,0,4,5,
	2,9,2,0,7,
	7,9,3,2,0
};

__global__ void compareAndReplace(int graf[N][N])
{
	int i = threadIdx.x;
	int k = threadIdx.y;
	for (int j = 0; j < N; j++)
	{
		if (graf[i][k] + graf[k][j] < graf[i][j])
			graf[i][j] = graf[i][k] + graf[k][j];
	}
}

int main(int argc, char** argv)
{
	int* gr;
	int sz = N * N * sizeof(int);
	hipMalloc(&gr, sz);
	hipMemcpy(gr, graf, size, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(N, N);
	compareAndReplace <<<numBlocks, threadsPerBlock >>>(A, B, C);
	hipMemcpy(graf, gr, size, hipMemcpyDeviceToHost);
	hipFree(gr);
	for (int i = 0; k < N; k++)
	{
		for (int j = 0; j < N; j++)
		{
			cout << graf[i][j] << " ";
		}
		cout << '\n';
	}
	return 0;
}
